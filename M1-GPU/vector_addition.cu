// This example demonstrates parallel floating point vector
// addition with a simple __global__ function.


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>


// this kernel computes the vector sum c = a + b
// each thread performs one pair-wise addition
__global__ void vector_add(const float *a,
                           const float *b,
                           float *c,
                           const size_t n)
{
  // compute the global element index this thread should process
  unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;

  // avoid accessing out of bounds elements
  if(i < n)
  {
    // sum elements
    c[i] = a[i] + b[i];
  }
}


int main(void)
{
  // create arrays of 1M elements
  const int num_elements = 1<<20;

  // compute the size of the arrays in bytes
  const int num_bytes = num_elements * sizeof(float);

  // points to host & device arrays
  float *device_array_a = 0;
  float *device_array_b = 0;
  float *device_array_c = 0;
  float *host_array_a   = 0;
  float *host_array_b   = 0;
  float *host_array_c   = 0;

  // malloc the host arrays
  host_array_a = (float*)malloc(num_bytes);
  host_array_b = (float*)malloc(num_bytes);
  host_array_c = (float*)malloc(num_bytes);

  // cudaMalloc the device arrays
  hipMalloc((void**)&device_array_a, num_bytes);
  hipMalloc((void**)&device_array_b, num_bytes);
  hipMalloc((void**)&device_array_c, num_bytes);

  // if any memory allocation failed, report an error message
  if(host_array_a == 0 || host_array_b == 0 || host_array_c == 0 ||
     device_array_a == 0 || device_array_b == 0 || device_array_c == 0)
  {
    printf("couldn't allocate memory\n");
    return 1;
  }

  // initialize host_array_a & host_array_b
  for(int i = 0; i < num_elements; ++i)
  {
    // make array a a linear ramp
    host_array_a[i] = (float)i;

    // make array b random
    host_array_b[i] = (float)rand() / RAND_MAX;
  }

  // copy arrays a & b to the device memory space
  hipMemcpy(device_array_a, host_array_a, num_bytes, hipMemcpyHostToDevice);
  hipMemcpy(device_array_b, host_array_b, num_bytes, hipMemcpyHostToDevice);

  // compute c = a + b on the device
  const size_t block_size = 256;
  size_t grid_size = num_elements / block_size;

  // deal with a possible partial final block
  if(num_elements % block_size) ++grid_size;

  // launch the kernel
  vector_add<<<grid_size, block_size>>>(device_array_a, device_array_b, device_array_c, num_elements);

  // copy the result back to the host memory space
  hipMemcpy(host_array_c, device_array_c, num_bytes, hipMemcpyDeviceToHost);

  // print out the first 10 results
  for(int i = 0; i < 10; ++i)
  {
    printf("result %d: %1.1f + %7.1f = %7.1f\n", i, host_array_a[i], host_array_b[i], host_array_c[i]);
  }

  // deallocate memory
  free(host_array_a);
  free(host_array_b);
  free(host_array_c);

  hipFree(device_array_a);
  hipFree(device_array_b);
  hipFree(device_array_c);
}
